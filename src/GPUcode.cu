__global__ void squared_kernel(int *in, int *out) {

  for (unsigned int i=0;i<blockDim.x;++i) {
    // /*const*/ unsigned int thread = threadIdx.x;
    out[threadIdx.x] = in[threadIdx.x] * in[threadIdx.x];
  }
};


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <iostream>
using namespace std;
int main(){
    cout<<"hello!"<<endl;
    return 0;
}
